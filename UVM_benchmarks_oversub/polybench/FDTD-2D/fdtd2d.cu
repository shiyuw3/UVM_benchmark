#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

/* Problem size */
// #define tmax 500
// #define NX 2048 * 6
// #define NY 2048 * 6

#define tmax 20

// NX=NY=26100: ~100%, 27000: ~105%.
#define NX 27000
#define NY 27000

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

#define ENABLE_CPU 0

#if ENABLE_CPU
void init_arrays(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz,
                 DATA_TYPE* _fict_gpu, DATA_TYPE* ex_gpu, DATA_TYPE* ey_gpu,
                 DATA_TYPE* hz_gpu) {
#else
void init_arrays(DATA_TYPE* _fict_gpu, DATA_TYPE* ex_gpu, DATA_TYPE* ey_gpu,
                 DATA_TYPE* hz_gpu) {
#endif
  int i, j;

  for (i = 0; i < tmax; i++) {
#if ENABLE_CPU
    _fict_[i] = (DATA_TYPE)i;
#endif
    _fict_gpu[i] = (DATA_TYPE)i;
  }

  for (i = 0; i < NX; i++) {
    for (j = 0; j < NY; j++) {
#if ENABLE_CPU
      ex[i * NY + j] = ((DATA_TYPE)i * (j + 1) + 1) / NX;
      ey[i * NY + j] = ((DATA_TYPE)(i - 1) * (j + 2) + 2) / NX;
      hz[i * NY + j] = ((DATA_TYPE)(i - 9) * (j + 4) + 3) / NX;
#endif
      ex_gpu[i * NY + j] = ((DATA_TYPE)i * (j + 1) + 1) / NX;
      ey_gpu[i * NY + j] = ((DATA_TYPE)(i - 1) *(j + 2) + 2) / NX;
      hz_gpu[i * NY + j] = ((DATA_TYPE)(i - 9) *(j + 4) + 3) / NX;
    }
  }
}

void runFdtd(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz) {
  int t, i, j;

  for (t = 0; t < tmax; t++) {
    for (j = 0; j < NY; j++) {
      ey[0 * NY + j] = _fict_[t];
    }

    for (i = 1; i < NX; i++) {
      for (j = 0; j < NY; j++) {
        ey[i * NY + j] = ey[i * NY + j] - 0.5 *
                             (hz[i * NY + j] - hz[(i - 1) * NY + j]);
      }
    }

    for (i = 0; i < NX; i++) {
      for (j = 1; j < NY; j++) {
        ex[i * (NY + 1) + j] = ex[i * (NY + 1) + j] - 0.5 *
                                   (hz[i * NY + j] - hz[i * NY + (j - 1)]);
      }
    }

    for (i = 0; i < NX; i++) {
      for (j = 0; j < NY; j++) {
        hz[i * NY + j] =
            hz[i * NY + j] - 0.7 * (ex[i * (NY + 1) + (j + 1)] -
                ex[i * (NY + 1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
      }
    }
  }
}

void compareResults(DATA_TYPE* hz1, DATA_TYPE* hz2) {
  int i, j, fail;
  fail = 0;

  for (i = 0; i < NX; i++) {
    for (j = 0; j < NY; j++) {
      if (percentDiff(hz1[i * NY + j], hz2[i * NY + j]) >
              PERCENT_DIFF_ERROR_THRESHOLD) {
        fail++;
      }
    }
  }

  // Print results
  printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f "
         "Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void GPU_argv_init() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
  printf("setting device %d with name %s\n", GPU_DEVICE,deviceProp.name);
  hipSetDevice(GPU_DEVICE);
}

__global__ void fdtd_step1_kernel(
    DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NX) && (j < NY)) {
    if (i == 0) {
      ey[i * NY + j] = _fict_[t];
    } else {
      ey[i * NY + j] = ey[i * NY + j] - 0.5f * (
                           hz[i * NY + j] - hz[(i - 1) * NY + j]);
    }
  }
}

__global__ void fdtd_step2_kernel(
    DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NX) && (j < NY) && (j > 0)) {
    ex[i * (NY + 1) + j] = ex[i * (NY + 1) + j] - 0.5f * (
                               hz[i * NY + j] - hz[i * NY + (j - 1)]);
  }
}

__global__ void fdtd_step3_kernel(
    DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if ((i < NX) && (j < NY)) {
    hz[i * NY + j] =
        hz[i * NY + j] - 0.7f * (ex[i * (NY + 1) + (j + 1)] -
            ex[i * (NY + 1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
  }
}

void fdtdCuda(DATA_TYPE* _fict_gpu, DATA_TYPE* ex_gpu, DATA_TYPE* ey_gpu,
              DATA_TYPE* hz_gpu) {
  double t_start, t_end;

  dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
  dim3 grid((size_t)ceil(((float)NY) / ((float)block.x)),
            (size_t)ceil(((float)NX) / ((float)block.y)));

  t_start = rtclock();

  for (int t = 0; t < tmax; t++) {
    fdtd_step1_kernel<<<grid, block>>>(_fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
    hipDeviceSynchronize();
    fdtd_step2_kernel<<<grid, block>>>(ex_gpu, ey_gpu, hz_gpu, t);
    hipDeviceSynchronize();
    fdtd_step3_kernel<<<grid, block>>>(ex_gpu, ey_gpu, hz_gpu, t);
    hipDeviceSynchronize();
  }

  t_end = rtclock();
  fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
}

int main() {
#if ENABLE_CPU
  DATA_TYPE* _fict_;
  DATA_TYPE* ex;
  DATA_TYPE* ey;
  DATA_TYPE* hz;

  _fict_ = (DATA_TYPE*)malloc(tmax * sizeof(DATA_TYPE));
  ex = (DATA_TYPE*)malloc(NX * (NY + 1) * sizeof(DATA_TYPE));
  ey = (DATA_TYPE*)malloc((NX + 1) * NY * sizeof(DATA_TYPE));
  hz = (DATA_TYPE*)malloc(NX * NY * sizeof(DATA_TYPE));
#endif

  DATA_TYPE *_fict_gpu;
  DATA_TYPE *ex_gpu;
  DATA_TYPE *ey_gpu;
  DATA_TYPE *hz_gpu;

  hipMallocManaged(&_fict_gpu, sizeof(DATA_TYPE) * tmax);
  hipMallocManaged(&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
  hipMallocManaged(&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
  hipMallocManaged(&hz_gpu, sizeof(DATA_TYPE) * NX * NY);

#if ENABLE_CPU
  init_arrays(_fict_, ex, ey, hz, _fict_gpu, ex_gpu, ey_gpu, hz_gpu);
#else
  init_arrays(_fict_gpu, ex_gpu, ey_gpu, hz_gpu);
#endif

  GPU_argv_init();
  fdtdCuda(_fict_gpu, ex_gpu, ey_gpu, hz_gpu);

#if ENABLE_CPU
  double t_start, t_end;
  t_start = rtclock();
  runFdtd(_fict_, ex, ey, hz);
  t_end = rtclock();
  fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

  compareResults(hz, hz_gpu);

  free(_fict_);
  free(ex);
  free(ey);
  free(hz);
#endif

  hipFree(_fict_gpu);
  hipFree(ex_gpu);
  hipFree(ey_gpu);
  hipFree(hz_gpu);
  return 0;
}
